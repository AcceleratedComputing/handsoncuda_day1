#include <stdio.h>
#include <stdlib.h>
#include "hip/hip_runtime.h"


__global__ void hello_kernel()
{
	printf("Hello World from Thread %d", threadIdx.x);
}


int main(int argc, char *argv[])
{

	dim3 blocksPerGrid(1, 1, 1);
	dim3 threadsPerBlock(10, 1, 1);
	hello_kernel<<<blocksPerGrid, threadsPerBlock>>>();
	
	return 0;
}